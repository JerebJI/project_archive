#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 1

#define S(dx,dy) ((cx+dx<0 || cy+dy<0 || cy+dx>=width || cy+dy>=height) ? 0 : imageIn[(cy+dy)*width+cx+dx])

__global__ void process(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height)
{
    // Izracunam koordinato niti
    int cx = blockIdx.x*blockDim.x + threadIdx.x;
    int cy = blockIdx.y*blockDim.y + threadIdx.y;
    
    // Prekinem, ce je nevaljavna koordinata
    if (cx>=width || cy>=height) return;

    // Izracunam potrebno
    int gx = -S(-1,-1)-2*S(0,-1)-S(1,-1)+S(-1,1)+2*S(0,1)+S(1,1);
    int gy = S(-1,-1)+2*S(-1,0)+S(-1,1)-S(1,-1)-2*S(1,0)-S(1,1);
    int r = sqrt((float)(gx*gx + gy*gy));
    
    // Zapisem rezultat
    imageOut[cy*width+cx] = (char)(r<255 ? r : 255);
}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("USAGE: sample input_image output_image\n");
        exit(EXIT_FAILURE);
    }
    
    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    cpp = COLOR_CHANNELS;

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    //printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    // Kot preizkus samo kopiramo vhodno sliko v izhodno
    memcpy(h_imageOut,h_imageIn,datasize);

    int bw = 32;
    int bh = 32;

    // Nastavimo organizacijo niti v 2D
    dim3 blockSize(bw, bh);
    dim3 gridSize(width/bw+1, height/bh+1);

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Rezervacija pomnilnika na napravi
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Uporabimo dogodke CUDA za merjenje casa
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Zacnem stopati
    hipEventRecord(start);

    // Kopiram sliko na napravo
    hipMemcpy(d_imageIn,h_imageIn,datasize,hipMemcpyHostToDevice);
    
    // Zazenemo scepec
    process<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height);
    getLastCudaError("process() execution failed\n");
    
    // Kopiram rezultat
    hipMemcpy(h_imageOut,d_imageOut,datasize,hipMemcpyDeviceToHost);
    
    // Koncam s stopanjem
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    // Izpisemo cas
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);
    printf("%f\n",milliseconds);

    // Zapisemo izhodno sliko v datoteko
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }

    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Sprostimo pomnilnik na napravi
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Pocistimo dogodke
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    // Sprostimo pomnilnik na gostitelju
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}